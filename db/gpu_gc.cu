#include "hip/hip_runtime.h"
//
// Created by ubuntu on 7/22/24.
//

#include <thrust/iterator/counting_iterator.h>

#include "gpu_gc.h"

void GPUGC::MallocMemory() {
  hipMalloc(&gpu_flags, max_num_log * max_num_log_item);
  hipMemset(gpu_flags, 1, max_num_log * max_num_log_item);

  hipMalloc(&invalid_count, max_num_log * sizeof(uint32_t));
  hipMemset(invalid_count, 0, max_num_log * sizeof(uint32_t));

  hipStreamCreate(&stream);
}

GPUGC::~GPUGC() {
  hipFree(gpu_flags);
  hipFree(invalid_count);
  hipStreamDestroy(stream);
}

__device__ bool CompareKey(const GPUKeyValue& key_value1,
                           const GPUKeyValue& key_value2) {
  for (int i = 0; i < keySize_; ++i) {
    if (key_value1.key[i] != key_value2.key[i]) {
      return false;
    }
  }
  return true;
}

__global__ void MarkInvalidKeysKernel(GPUKeyValue* key_values_d,
                                      uint8_t* gpu_flags,
                                      uint32_t max_num_log_item,
                                      uint32_t var_key_value_size,
                                      uint32_t* invalid_count, size_t n) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid > 0 && tid < n) {
    if (CompareKey(key_values_d[tid], key_values_d[tid - 1])) {
      uint32_t vlog_num = GPUDecodeFixed32(key_values_d[tid].value);
      uint32_t invalid_pos = GPUDecodeFixed32(key_values_d[tid].value + 4);
      uint32_t idx = (vlog_num - 1) * max_num_log_item +
                     (invalid_pos - 12) / var_key_value_size;
      gpu_flags[idx] = 0;  // 索引从0开始
      // 该vlog的无效KV对数量+1，必须原子操作
      atomicAdd(&invalid_count[vlog_num - 1], 1);
    }
  }
}

void GPUGC::Mark(GPUKeyValue* key_values_d, size_t n) {
  // 计算块和线程数
  size_t threadsPerBlock = 1024;
  size_t blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

  MarkInvalidKeysKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
      key_values_d, gpu_flags, max_num_log_item,
      leveldb::my_stats.var_key_value_size + 12, invalid_count, n);
  CHECK(hipStreamSynchronize(stream));
}

__global__ void TriggerGCKernel(const uint32_t* invalid_count,
                                uint32_t* vlog_num_d,
                                uint32_t* curr_invalid_count,
                                uint32_t max_num_log,
                                uint32_t clean_threshold) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= max_num_log) return;

  // 使用 atomicCAS 确保只有第一个线程进入
  if (invalid_count[tid] >= clean_threshold) {
    if (atomicCAS(vlog_num_d, 0, tid + 1) == 0) {  // vlog_num_d 初始值应为 0
      // 只有第一个线程成功更新 vlog_num_d，才能进入这里
      *curr_invalid_count = invalid_count[tid];
    }
    return;
  }
}

__global__ void Adjustment(const uint8_t* flags, uint32_t max_num_log_item,
                           uint32_t* count) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= max_num_log_item) return;
  if (flags[tid] == 0) atomicAdd(count, 1);
}

bool GPUGC::TriggerGC() {
  uint32_t* vlog_num_d;
  uint32_t* curr_invalid_count;
  hipMalloc(&vlog_num_d, sizeof(uint32_t));
  hipMemset(vlog_num_d, 0, sizeof(uint32_t));
  hipMalloc(&curr_invalid_count, sizeof(uint32_t));
  hipMemset(curr_invalid_count, 0, sizeof(uint32_t));

  size_t threadsPerBlock = 256;
  size_t blocksPerGrid = (max_num_log + threadsPerBlock - 1) / threadsPerBlock;
  TriggerGCKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
      invalid_count, vlog_num_d, curr_invalid_count, max_num_log,
      leveldb::my_stats.clean_threshold);
  CHECK(hipStreamSynchronize(stream));

  hipMemcpy(&triggered_vlog_num, vlog_num_d, sizeof(uint32_t),
             hipMemcpyDeviceToHost);

  if (triggered_vlog_num > 0) {
    hipMemcpy(&triggered_invalid_count, curr_invalid_count, sizeof(uint32_t),
               hipMemcpyDeviceToHost);

    printf("GC is triggered, vlog num: %d, count: %d\n", triggered_vlog_num,
           triggered_invalid_count);
    return true;
  }
  return false;
}

__global__ void GPUGCKernel(char* vlog_d, char* output_d, uint8_t* flags,
                            uint32_t max_num_log_item,
                            uint32_t triggered_vlog_num, uint32_t* global_count,
                            uint32_t var_key_value_size) {
  uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid >= max_num_log_item) return;

  flags = flags + (triggered_vlog_num - 1) * max_num_log_item;
  if (flags[tid] == 0) return;

  uint32_t index = atomicAdd(global_count, 1);
  memcpy(output_d + index * var_key_value_size,
         vlog_d + tid * var_key_value_size, var_key_value_size);
}

void GPUGC::BeginGPUGC(const char* vlog, size_t vlog_size, char** output,
                       size_t& output_size) {
  char* vlog_d;
  hipMalloc(&vlog_d, vlog_size);
  hipMemcpy(vlog_d, vlog, vlog_size, hipMemcpyHostToDevice);

  uint32_t* global_count;
  hipMalloc(&global_count, max_num_log_item * sizeof(uint32_t));
  hipMemset(global_count, 0, max_num_log_item * sizeof(uint32_t));

  output_size = vlog_size - triggered_invalid_count *
                                (leveldb::my_stats.var_key_value_size + 12);
  char* output_d;
  hipMalloc(&output_d, output_size);

  // 计算块和线程数
  size_t threadsPerBlock = 1024;
  size_t blocksPerGrid =
      (max_num_log_item + threadsPerBlock - 1) / threadsPerBlock;

  GPUGCKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
      vlog_d, output_d, gpu_flags, max_num_log_item, triggered_vlog_num,
      global_count, leveldb::my_stats.var_key_value_size + 12);

  CHECK(hipStreamSynchronize(stream));

  *output = new char[output_size];
  hipMemcpy(*output, output_d, output_size, hipMemcpyDeviceToHost);

  hipFree(output_d);
  hipFree(global_count);
  hipFree(vlog_d);
}

__global__ void GPUGCOptimizedKernel(char* vlog_d, char* output_d,
                                     const uint8_t* flags,
                                     uint32_t var_key_value_size,
                                     uint32_t total_thread_num,
                                     uint32_t process_num_per_thread,
                                     uint32_t* global_count) {
  uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid >= total_thread_num) return;

  uint32_t count;
  for (uint32_t idx = tid * process_num_per_thread;
       idx < (tid + 1) * process_num_per_thread; ++idx) {
    if (flags[idx] == 1) {
      count = atomicAdd(global_count, 1);
      memcpy(output_d + count * var_key_value_size,
             vlog_d + idx * var_key_value_size, var_key_value_size);
    }
  }
}

void GPUGC::BeginGPUGCOptimized(const char* vlog, size_t vlog_size,
                                char** output) {
  auto start_time = std::chrono::high_resolution_clock::now();
  char* vlog_d;
  hipMalloc((void**)&vlog_d, vlog_size);
  hipMemcpy(vlog_d, vlog, vlog_size, hipMemcpyHostToDevice);
  auto end_time = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(
      end_time - start_time);
  leveldb::my_stats.data_transfer_time += duration.count();

  uint32_t process_num_per_thread = 100;
  uint32_t total_thread_num =
      leveldb::my_stats.max_num_log_item / process_num_per_thread;

  uint32_t* global_count;
  hipMalloc(&global_count, sizeof(uint32_t));
  hipMemset(global_count, 0, sizeof(uint32_t));

  uint32_t* flag_count_d;
  hipMalloc(&flag_count_d, sizeof(uint32_t));
  hipMemset(flag_count_d, 0, sizeof(uint32_t));
  size_t block = 256;
  size_t grid = (max_num_log_item + block - 1) / block;
  Adjustment<<<grid, block, 0, stream>>>(
      gpu_flags + (triggered_vlog_num - 1) * max_num_log_item, max_num_log_item,
      flag_count_d);
  CHECK(hipStreamSynchronize(stream));
  uint32_t flag_count;
  hipMemcpy(&flag_count, flag_count_d, sizeof(uint32_t),
             hipMemcpyDeviceToHost);
  //  printf("flag count: %u\n", flag_count);

  triggered_invalid_count = flag_count;

  size_t output_size =
      vlog_size - flag_count * (leveldb::my_stats.var_key_value_size + 12);

  char* output_d;
  hipMalloc(&output_d, output_size);

  // 计算块和线程数
  size_t threadsPerBlock = 1024;
  size_t blocksPerGrid =
      (total_thread_num + threadsPerBlock - 1) / threadsPerBlock;
  GPUGCOptimizedKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
      vlog_d, output_d, gpu_flags + (triggered_vlog_num - 1) * max_num_log_item,
      leveldb::my_stats.var_key_value_size + 12, total_thread_num,
      process_num_per_thread, global_count);
  CHECK(hipStreamSynchronize(stream));

  *output = new char[output_size];
  hipMemcpy(*output, output_d, output_size, hipMemcpyDeviceToHost);

  hipFree(output_d);
  hipFree(vlog_d);
  hipFree(global_count);
}

void GPUGC::CleanGC() {
  hipMemset(&invalid_count[triggered_vlog_num - 1], 0, sizeof(uint32_t));
  hipMemset(gpu_flags + (triggered_vlog_num - 1) * max_num_log_item, 1,
             max_num_log_item);
  triggered_vlog_num = 0;
  triggered_invalid_count = 0;
}
